#include <iostream>
#include <hip/hip_runtime.h>

const int N = 1 << 20; // 1 million elements

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory on CPU
    a = new float[N];
    b = new float[N];
    out = new float[N];

    // Initialize arrays
    for(int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memory on GPU
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Copy data from CPU to GPU
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Launch kernel
    vector_add<<<(N + 255) / 256, 256>>>(d_out, d_a, d_b, N);

    // Copy results back to CPU
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Check results
    for(int i = 0; i < N; i++) {
        if(a[i] + b[i] != out[i]) {
            std::cerr << "Error: " << a[i] + b[i] << " != " << out[i] << std::endl;
            delete[] a;
            delete[] b;
            delete[] out;
            return -1;
        }
    }

    std::cout << "Vector addition successful!" << std::endl;

    delete[] a;
    delete[] b;
    delete[] out;

    return 0;
}

